#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "args.h"
#include "boundary.h"
#include "data.h"
#include "setup.h"
#include "vtk.h"

void CUDAErrorCheck() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
        exit(0);
    }
}


__global__ void cuda_comp_accel(int * d_part_neighbour_list, double * d_part_x, double * d_part_y,
								double * d_part_ax, double * d_part_ay, int * d_part_i, int * d_part_j, double * d_pot_energy_arr,
								double cell_size, double r_cut_off, double r_cut_off_2,
								double Duc, double Uc, int y, int num_particles, int num_part_per_dim) {
	int p = blockIdx.x * blockDim.x + threadIdx.x;
	
	int i = d_part_i[p];
	int j = d_part_j[p];

	// printf("p %d\n", p);

	for (int n = 0; n < 2*num_part_per_dim*num_part_per_dim; n++) {
		int q = d_part_neighbour_list[(p*2*num_part_per_dim*num_part_per_dim) + n];
		// printf("q %d\n", q);
		
		if (q > -1) {

			int iq = d_part_i[q];
			int jq = d_part_j[q];
			// since particles are stored relative to their cell, calculate the
			// actual x and y coordinates.

			double p_real_x = ((i-1) * cell_size) + d_part_x[p];
			double p_real_y = ((j-1) * cell_size) + d_part_y[p];
			double q_real_x = ((iq-1) * cell_size) + d_part_x[q];
			double q_real_y = ((jq-1) * cell_size) + d_part_y[q];
			
			// calculate distance in x and y, then absolute distance
			double dx = p_real_x - q_real_x;
			double dy = p_real_y - q_real_y;
			double r_2 = dx*dx + dy*dy;
			
			// if distance less than cut off, calculate force and 
			// use this to calculate acceleration in each dimension
			// calculate potential energy of each particle at the same time
			if (r_2 < r_cut_off_2) {
				double r_2_inv = 1.0 / r_2;
				double r_6_inv = r_2_inv * r_2_inv * r_2_inv;
				
				double f = (48.0 * r_2_inv * r_6_inv * (r_6_inv - 0.5));

				d_part_ax[p] += f*dx;

				d_part_ay[p] += f*dy;
				d_pot_energy_arr[p] += 4.0 * r_6_inv * (r_6_inv - 1.0) - Uc - Duc * (sqrt(r_2) - r_cut_off);
			}
		}
	}
}


/**
 * @brief This routine calculates the acceleration felt by each particle based on evaluating the Lennard-Jones 
 *        potential with its neighbours. It only evaluates particles within a cut-off radius, and uses cells to 
 *        reduce the search space. It also calculates the potential energy of the system. 
 * 
 * @return double The potential energy
 */
double comp_accel(double * pot_energy_arr, double * d_pot_energy_arr) {
	// zero acceleration for every particle
	for (int p = 0; p < num_particles; p++) {
		particles.ax[p] = 0.0;
		particles.ay[p] = 0.0;
		particles.num_neighbours[p] = 0;
	}

	for (int m = 0; m < num_particles*2*num_part_per_dim*num_part_per_dim; m++) {
		part_neighbour_list[m] = -1;
	}

	double pot_energy = 0.0;
	hipMemset(d_pot_energy_arr, 0.0, sizeof(double) * num_particles);

	// Compare each particle with all particles in the 9 cells
	for (int i = 1; i < x+1; i++) {
		for (int j = 1; j < y+1; j++) {
			for (int k = 0; k < cells[i][j].count; k++) {
				int p = cells[i][j].part_ids[k];
				// Compare each particle with all particles in the 9 cells
				for (int a = -1; a <= 1; a++) {
					for (int b = -1; b <= 1; b++) {
						for (int l = 0; l < cells[i+a][j+b].count; l++) {
							int q = cells[i+a][j+b].part_ids[l];
							if (p == q) {
								continue;
							}
							part_neighbour_list[(p*2*num_part_per_dim*num_part_per_dim) + particles.num_neighbours[p]] = q;
							particles.num_neighbours[p]++;
						}
					}
				}
			}
		}
	}
	hipMemcpy(d_part_neighbour_list, part_neighbour_list, sizeof(int)*num_particles*2*num_part_per_dim*num_part_per_dim, hipMemcpyHostToDevice);
	hipMemcpy(d_part_ax, particles.ax, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	hipMemcpy(d_part_ay, particles.ay, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	CUDAErrorCheck();
	hipDeviceSynchronize();

	int block_size = 256;
	int grid_size = num_particles / block_size;
	cuda_comp_accel<<<grid_size,block_size>>>(d_part_neighbour_list, d_part_x, d_part_y, d_part_ax, d_part_ay,
												d_part_i, d_part_j, d_pot_energy_arr, cell_size,
												r_cut_off, r_cut_off_2, Duc, Uc, y, num_particles, num_part_per_dim);
	CUDAErrorCheck();
	hipMemcpy(pot_energy_arr, d_pot_energy_arr, sizeof(double) * num_particles, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for (int p = 0; p < num_particles; p++) {
		pot_energy += pot_energy_arr[p];
	}

	// return the average potential energy (i.e. sum / number)
	return pot_energy / num_particles;
}

/**
 * @brief This routine updates the velocity of each particle for half a time step and then 
 *        moves the particle for a whole time step
 * 
 */
__global__ void move_particles(double * d_part_vx, double * d_part_vy,
								double * d_part_x, double * d_part_y,
								double * d_part_ax, double * d_part_ay,
								int num_particles, double dt, double dth) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// move all particles half a time step
	if (tid < num_particles) {
		// update velocity to obtain v(t + Dt/2)
		d_part_vx[tid] += dth * d_part_ax[tid];
		d_part_vy[tid] += dth * d_part_ay[tid];

		// update particle coordinates to p(t + Dt) (scaled to the cell_size)
		d_part_x[tid] += (dt * d_part_vx[tid]);
		d_part_y[tid] += (dt * d_part_vy[tid]);
	}
}

/**
 * @brief This routine updates the cell lists. If a particles coordinates are not within a cell
 *        any more, this function calculates the cell it should be in and performs the move.
 *        If a particle moves more than 1 cell in any direction, this indicates poor settings
 *        and therefore an error is generated.
 * 
 */
void update_cells() {
	// move particles that need to move cell lists
	for (int i = 1; i < x+1; i++) {
		for (int j = 1; j < y+1; j++) {
			// we have to store the next particle here, as the remove/add at the end may be destructive
			
			int cell_count = cells[i][j].count;
			int * cell_part_ids = cells[i][j].part_ids;
			for (int k = 0; k < cell_count; k++) {
				int p = cell_part_ids[k];

				// if a particles x or y value is greater than the cell size or less than 0, it must have moved cell
				// do a quick check to make sure its not moved 2 cells (since this means our time step is too large, or something else is going wrong)
				if ((particles.x[p] < 0.0) | (particles.x[p] >= cell_size) | (particles.y[p] < 0.0) | (particles.y[p] >= cell_size)) {
					if ((particles.x[p] < (-cell_size)) || (particles.x[p] >= (2*cell_size)) || (particles.y[p] < (-cell_size)) || (particles.y[p] >= (2*cell_size))) {
						fprintf(stderr, "A particle has moved more than one cell!\n");
						exit(1);
					}

					// work out whether we've moved a cell in the x and the y dimension
					int x_shift = (particles.x[p] < 0.0) ? -1 : (particles.x[p] >= cell_size) ? +1 : 0;
					int y_shift = (particles.y[p] < 0.0) ? -1 : (particles.y[p] >= cell_size) ? +1 : 0;
					
					// the new i and j are +/- 1 in each dimension,
					// but if that means we go out of simulation bounds, wrap it to x and 1
					int new_i = i+x_shift;
					if (new_i == 0) { new_i = x; }
					if (new_i == x+1) { new_i = 1; }
					int new_j = j+y_shift;
					if (new_j == 0) { new_j = y; }
					if (new_j == y+1) { new_j = 1; }
					// update x and y coordinates (i.e. remove the additional cell size)
					particles.x[p] = particles.x[p] + (x_shift * -cell_size);
					particles.y[p] = particles.y[p] + (y_shift * -cell_size);

					// remove the particle from its current cell list, then add it to the new cell list
					remove_particle(&(cells[i][j]), k);
					add_particle(&(cells[new_i][new_j]), p, new_i, new_j);
				}
			}
		}
	}
}

/**
 * @brief This updates the velocity of particles for the whole time step (i.e. adds the acceleration for another
 *        half step, since its already done half a time step in the move_particles routine). Additionally, this
 *        function calculated the kinetic energy of the system.
 * 
 * @return double The kinetic energy
 */
double update_velocity() {
	double kinetic_energy = 0.0;

	for (int p = 0; p < num_particles; p++) {
		// update velocity again by half time to obtain v(t + Dt)
		particles.vx[p] += dth * particles.ax[p];
		particles.vy[p] += dth * particles.ay[p];

		// calculate the kinetic energy by adding up the squares of the velocities in each dim
		kinetic_energy += (particles.vx[p] * particles.vx[p]) + (particles.vy[p] * particles.vy[p]);
	}

	// KE = (1/2)mv^2
	kinetic_energy *= (0.5 / num_particles);
	return kinetic_energy;
}

/**
 * @brief This is the main routine that sets up the problem space and then drives the solving routines.
 * 
 * @param argc The number of arguments passed to the program
 * @param argv An array of the arguments passed to the program
 * @return int The exit code of the application
 */
int main(int argc, char *argv[]) {
	hipEvent_t start, stop;
    float gpu_time;

	// Set default parameters
	set_defaults();
	// parse the arguments
	parse_args(argc, argv);
	// call set up to update defaults
	setup();

	if (verbose) print_opts();
	
	// create events for time profiling
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the event timer
    hipEventRecord(start, 0);

	// set up problem
	problem_setup();

	// apply boundary condition (i.e. update pointers on the boundarys to loop periodically)
	apply_boundary();

	double * d_pot_energy_arr;
	double * pot_energy_arr = (double *) malloc(sizeof(double) * num_particles);
	
	hipMalloc((void **) &d_part_i, sizeof(int) * num_particles);
	hipMalloc((void **) &d_part_j, sizeof(int) * num_particles);
	hipMalloc((void **) &d_part_x, sizeof(double) * num_particles);
	hipMalloc((void **) &d_part_y, sizeof(double) * num_particles);
	hipMalloc((void **) &d_part_ax, sizeof(double) * num_particles);
	hipMalloc((void **) &d_part_ay, sizeof(double) * num_particles);
	hipMalloc((void **) &d_part_vx, sizeof(double) * num_particles);
	hipMalloc((void **) &d_part_vy, sizeof(double) * num_particles);
	hipMalloc((void **) &d_pot_energy_arr, sizeof(double) * num_particles);
	hipMalloc((void **) &d_part_neighbour_list, sizeof(int) * num_particles * (2*num_part_per_dim*num_part_per_dim));
	CUDAErrorCheck();

	hipMemcpy(d_part_i, particles.cell_i, sizeof(int)*num_particles, hipMemcpyHostToDevice);
	hipMemcpy(d_part_j, particles.cell_j, sizeof(int)*num_particles, hipMemcpyHostToDevice);
	hipMemcpy(d_part_x, particles.x, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	hipMemcpy(d_part_y, particles.y, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	hipMemcpy(d_part_vx, particles.vx, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	hipMemcpy(d_part_vy, particles.vy, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	comp_accel(pot_energy_arr, d_pot_energy_arr);

	hipMemcpy(particles.cell_i, d_part_i, sizeof(int)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.cell_j, d_part_j, sizeof(int)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.x, d_part_x, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.y, d_part_y, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.ax, d_part_ax, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.ay, d_part_ay, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.vx, d_part_vx, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
	hipMemcpy(particles.vy, d_part_vy, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	double potential_energy = 0.0;
	double kinetic_energy = 0.0;

	int iters = 0;
	double t;
	for (t = 0.0; t < t_end; t+=dt, iters++) {
		// move particles half a time step
		hipMemcpy(d_part_i, particles.cell_i, sizeof(int)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_j, particles.cell_j, sizeof(int)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_x, particles.x, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_y, particles.y, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_ax, particles.ax, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_ay, particles.ay, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_vx, particles.vx, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_vy, particles.vy, sizeof(double)*num_particles, hipMemcpyHostToDevice);
	    hipDeviceSynchronize();
		
		int block_size = 256;
		int grid_size = num_particles / block_size;
		move_particles<<<grid_size,block_size>>>(d_part_x, d_part_y, d_part_ax, d_part_ay, d_part_vx, d_part_vy, num_particles, dt, dth);
		
		hipMemcpy(particles.cell_i, d_part_i, sizeof(int)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.cell_j, d_part_j, sizeof(int)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.x, d_part_x, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.y, d_part_y, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.ax, d_part_ax, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.ay, d_part_ay, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.vx, d_part_vx, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.vy, d_part_vy, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
    	hipDeviceSynchronize();

		// update cell lists (i.e. move any particles between cell lists if required)
		update_cells();

		// update pointers (because the previous operation might break boundary cell lists)
		apply_boundary();
		
		// compute acceleration for each particle and calculate potential energy
		hipMemcpy(d_part_i, particles.cell_i, sizeof(int)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_j, particles.cell_j, sizeof(int)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_x, particles.x, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_y, particles.y, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_ax, particles.ax, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipMemcpy(d_part_ay, particles.ay, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		// hipMemcpy(d_part_vx, particles.vx, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		// hipMemcpy(d_part_vy, particles.vy, sizeof(double)*num_particles, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		potential_energy = comp_accel(pot_energy_arr, d_pot_energy_arr);

		hipMemcpy(particles.cell_i, d_part_i, sizeof(int)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.cell_j, d_part_j, sizeof(int)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.x, d_part_x, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.y, d_part_y, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.ax, d_part_ax, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		hipMemcpy(particles.ay, d_part_ay, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		// hipMemcpy(particles.vx, d_part_vx, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
		// hipMemcpy(particles.vy, d_part_vy, sizeof(double)*num_particles, hipMemcpyDeviceToHost);
    	hipDeviceSynchronize();

		// update velocity based on the acceleration and calculate the kinetic energy
		kinetic_energy = update_velocity();
	
		if (iters % output_freq == 0) {
			// calculate temperature and total energy
			double total_energy = kinetic_energy + potential_energy;
			double temp = kinetic_energy * 2.0 / 3.0;

			printf("Step %8d, Time: %14.8e (dt: %14.8e), Total energy: %14.8e (p:%14.8e,k:%14.8e), Temp: %14.8e\n", iters, t+dt, dt, total_energy, potential_energy, kinetic_energy, temp);
 
			// if output is enabled and checkpointing is enabled, write out
            if ((!no_output) && (enable_checkpoints))
                write_checkpoint(iters, t+dt);
		}
	}

	// calculate the final energy and write out a final status message
	double final_energy = kinetic_energy + potential_energy;
	printf("Step %8d, Time: %14.8e, Final energy: %14.8e\n", iters, t, final_energy);
    printf("Simulation complete.\n");

	// if output is enabled, write the mesh file and the final state
	if (!no_output) {
		write_mesh();
		write_result(iters, t);
	}

	return 0;
}

